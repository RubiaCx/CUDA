
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <hipblas.h>
#include <iostream>
#include <random>

void generate_random_value_float(float *result, int size, float lower_bound, float upper_bound)
{
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dist(lower_bound, upper_bound);

    for (int i = 0; i < size; ++i)
    {
        result[i] = dist(gen);
    }
}

template <int BLOCK_SIZE> 
__global__ void reduce(float *d_in, float *d_out)
{
    __shared__ float sdata[BLOCK_SIZE];
    // each thread loads one element from global memory to shared mem
    unsigned int i = blockIdx.x * blockDim.x*2 + threadIdx.x; // HERE
    unsigned int tid = threadIdx.x;
    sdata[tid] = d_in[i] + d_in[i + blockDim.x];
    __syncthreads();

    // do reduction in shared mem
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    // write result for this block to global mem
    if (tid == 0)
        d_out[blockIdx.x] = sdata[tid];
}

int main(int argc, char **argv)
{
    const int TEST_TIMES = 10;
    const int N = 32 * 1024 * 1024;
    const int BLOCK_SIZE = 256;

    int32_t BLOCK_NUM = (N + BLOCK_SIZE - 1) / BLOCK_SIZE / 2; // AND HERE
    dim3 grid(BLOCK_NUM, 1);
    dim3 block(BLOCK_SIZE, 1);

    float *h_i_data = (float *)malloc(N * sizeof(float));
    generate_random_value_float(h_i_data, N, 1.0, 2.0);
    float *h_o_data = (float *)malloc(N / BLOCK_SIZE * sizeof(float));
    float *d_i_data, *d_o_data;
    hipMalloc(&d_i_data, N * sizeof(float));
    hipMalloc(&d_o_data, N / BLOCK_SIZE * sizeof(float));

    hipMemcpy(d_i_data, h_i_data, N * sizeof(float), hipMemcpyHostToDevice);

    // events for timing
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    float ms;

    // ------------
    // generate answer
    // ------------
    float answer = 0.0f;
    for (int i = 0; i < N; ++i)
    {
        answer += h_i_data[i];
    }

    // ------------
    // time kernels
    // ------------

    //  warm up
    reduce<BLOCK_SIZE><<<grid, block>>>(d_i_data, d_o_data);

    hipEventRecord(startEvent, 0);
    for (int i = 0; i < TEST_TIMES; i++)
        reduce<BLOCK_SIZE><<<grid, block>>>(d_i_data, d_o_data);
    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    hipEventElapsedTime(&ms, startEvent, stopEvent);
    hipMemcpy(h_o_data, d_o_data, BLOCK_NUM * sizeof(float), hipMemcpyDeviceToHost);

    float my_answer = 0;
    for (int i = 0; i < BLOCK_NUM; i++)
    {
        my_answer += h_o_data[i * BLOCK_SIZE];
    }
    if ((my_answer - answer) < 1e-6) // 允许小于1e-6的误差 if (result[i] != answer[i])
    {
        printf("Time = %.6lf ms\n", ms / TEST_TIMES); // Time = 0.790528 ms
    }

    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);
    hipFree(d_i_data);
    hipFree(d_o_data);
    free(h_i_data);
    free(h_o_data);

    return 0;
}
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <random>
#include <math.h>
#include <chrono>
#include "../include/utils.h"
#include "../include/warp.cuh"
#include "../include/error.cuh"
// CUDA runtime
#include <hip/hip_runtime.h>
#include <hipblas.h>

// cal offset from row col and ld , in row-major matrix, ld is the width of the matrix
#define OFFSET(row, col, ld) ((row) * (ld) + (col))

// transfer float4
#define FETCH_FLOAT4(pointer) (reinterpret_cast<float4 *>(&(pointer))[0])

void cpuSgemv(float *A, float *x, float *y, int M, int N)
{
    for (int i = 0; i < M; i++)
    {
        float sum = 0.0;
        for (int j = 0; j < N; j++)
        {
            sum += A[i * N + j] * x[j];
        }
        y[i] = sum;
    }
}

// x = blockIdx.x * blockDim.x + threadIdx.x;
// y = blockIdx.y * blockDim.y + threadIdx.y
// N = 16 每个warp处理2行
template <const int WARP_SIZE, const int ROW_PER_WARP, class value_t>
__global__ void Sgemv_kernel_n16(value_t *__restrict__ A, value_t *__restrict__ x, value_t *__restrict__ y,
                                 const int M, const int N)
{
    int tx = threadIdx.x;  // 每个线程
    int warpRowBase = blockIdx.x * ROW_PER_WARP;  // 每个block处理ROW_PER_WARP行，blockIdx.x * ROW_PER_WARP 是第一行的索引

    // 处理第一行和第二行
    for (int row = warpRowBase; row < min(warpRowBase + ROW_PER_WARP, M); ++row) {
        if (tx < N) 
        {
            float res = A[row * N + tx] * x[tx];
            // 在warp内进行归约求和
            res = warpReduceSum<value_t>(res, WARP_SIZE);
            // 只有第一个线程写回结果到全局内存
            if (tx == 0)
                y[row] = res;
        }
    }

}

// N = 32，每个warp处理1行
template <const int WARP_SIZE, class value_t>
__global__ void Sgemv_kernel_n32(value_t *__restrict__ A, value_t *__restrict__ x, value_t *__restrict__ y,
                                 const int M, const int N)
{
    int tx = threadIdx.x;  // 每个线程处理一列
    int row = blockIdx.x;  // 每个块处理一行

    if (row < M)
    {
        value_t res = 0;
        // 每个线程处理一列中的元素
        res += A[row * N + tx] * x[tx];
        
        // 对 warp 内的结果进行归约求和
        res = warpReduceSum<value_t>(res, WARP_SIZE);

        // 只有第一个线程写回结果到全局内存
        if (tx == 0)
            y[row] = res;
    }
}

// N = 128，每个warp处理1行，向量化指令，即一个指令处理4个元素
template <const int WARP_SIZE, class value_t>
__global__ void Sgemv_kernel_n128(value_t *__restrict__ A, value_t *__restrict__ x, value_t *__restrict__ y,
                                  const int M, const int N)
{
    int tx = threadIdx.x;  
    int row = blockIdx.x;  // 每个块处理一行

    if (row < M)
    {
        value_t res = 0;
        int numVectorPerRow = N / 4; // 因为使用 float4，每行有 N/4 个向量
        int numVectorsPerThread = numVectorPerRow / WARP_SIZE;  // 计算每个线程应处理多少向量
        int vectorIndexOffset = tx * numVectorsPerThread;       // 每个线程处理向量的起始索引

        // 确保不越界
        int maxVectorIndex = min((tx + 1) * numVectorsPerThread, numVectorPerRow);

        for (int i = vectorIndexOffset; i < maxVectorIndex; i++)
        {
            int col = i * 4;  // 每个向量对应四个列
            float4 vecA = reinterpret_cast<float4*>(&A[row * N + col])[0];
            float4 vecX = reinterpret_cast<float4*>(&x[col])[0];
            res += vecA.x * vecX.x + vecA.y * vecX.y + vecA.z * vecX.z + vecA.w * vecX.w;
        }

        res = warpReduceSum<value_t>(res, WARP_SIZE);

        // Write the result to global memory
        if (tx == 0)
            y[row] = res;
    }
}


template <class value_t>
__device__ value_t kahanSum(value_t *data, int N)
{
    value_t sum = 0.0;
    value_t c = 0.0; // 一个运行时小的误差补偿变量
    for (int i = 0; i < N; ++i)
    {
        value_t y = data[i] - c;
        value_t t = sum + y;
        c = (t - sum) - y;
        sum = t;
    }
    return sum;
}

template <const int WARP_SIZE, const int ROW_PER_WARP, class value_t>
__global__ void Sgemv_kernel_Kahan(value_t *__restrict__ A, value_t *__restrict__ x, value_t *__restrict__ y,
                                   const int M, const int N)
{
    extern __shared__ value_t shared_data[];

    int bx = blockIdx.x;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int laneId = tx % WARP_SIZE;
    int current_warp_row = (blockDim.y * bx + ty) * ROW_PER_WARP;
    const int sub_WARP_SIZE = WARP_SIZE / ROW_PER_WARP;
    int kLaneId = laneId % sub_WARP_SIZE;
    int current_thread_row = current_warp_row + laneId / sub_WARP_SIZE;

    if (current_thread_row < M)
    {
        int col = kLaneId;
        while (col < N)
        {
            shared_data[threadIdx.x * N + col] = A[current_thread_row * N + col] * x[col];
            col += sub_WARP_SIZE;
        }
        __syncthreads(); // 确保所有数据都写入shared memory

        if (kLaneId == 0)
        { // 使用第一个线程来执行Kahan求和
            y[current_thread_row] = kahanSum<value_t>(&shared_data[threadIdx.x * N], N);
        }
    }
}

int main(int argc, char **argv)
{
    if (argc != 3)
    {
        printf("usage: ./main [M] [N]\n");
        exit(0);
    }
    size_t M = atoi(argv[1]);
    size_t N = atoi(argv[2]);

    size_t bytes_A = sizeof(float) * M * N;
    size_t bytes_x = sizeof(float) * N;
    size_t bytes_y = sizeof(float) * M;
    float *h_A = (float *)malloc(bytes_A);
    float *h_x = (float *)malloc(bytes_x);
    float *h_y = (float *)malloc(bytes_y);
    float *h_y_gpu = (float *)malloc(bytes_y);
    float *h_y_cpu = (float *)malloc(bytes_y);
    float *h_y_api = (float *)malloc(bytes_y);
    float *d_A;
    float *d_x;
    float *d_y;
    generate_random_value_float(h_A, M * N, 0.0, 1.0);
    generate_random_value_float(h_x, N, 0.0, 1.0);
    memset(h_y, 0, M * sizeof(float));
    memset(h_y_gpu, 0, M * sizeof(float));
    memset(h_y_cpu, 0, M * sizeof(float));
    memset(h_y_api, 0, M * sizeof(float));

    // // 输出前10个元素以验证
    // for (int i = 0; i < 10; ++i) {
    //     std::cout << h_A[i] << " ";
    // }
    // std::cout << std::endl;

    int nIter = 1;
    cpuSgemv(h_A, h_x, h_y_cpu, M, N);
    auto start_cpu = std::chrono::high_resolution_clock::now();
    for (int run = 0; run < nIter; run++)
    {
        cpuSgemv(h_A, h_x, h_y_cpu, M, N);
    }
    auto end_cpu = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> cpu_time = end_cpu - start_cpu;
    printf("CPU time: %.5f seconds\n", cpu_time.count() / nIter);
    // for (int i = 0; i < M; ++i) {
    //     std::cout << h_y_cpu[i] << " ";
    // }
    // std::cout << std::endl;
    const int WARP_SIZE = 32;
    const int ROW_PER_WARP = 2;

    float gpu_time = 0;
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));
    CHECK_CUDA(hipMalloc(&d_A, bytes_A));
    CHECK_CUDA(hipMalloc(&d_x, bytes_x));
    CHECK_CUDA(hipMalloc(&d_y, bytes_y));
    CHECK_CUDA(hipMemcpy(d_A, h_A, bytes_A, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_x, h_x, bytes_x, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_y, h_y, bytes_y, hipMemcpyHostToDevice));
    if(N <= 16) {
        dim3 dimGrid((M + ROW_PER_WARP - 1) / ROW_PER_WARP);  // 确保所有行都被处理，每个块处理 ROW_PER_WARP 行
        dim3 dimBlock(WARP_SIZE);  // 每行分配 WARP_SIZE / ROW_PER_WARP 个线程，每块处理 ROW_PER_WARP 行
        Sgemv_kernel_n16<WARP_SIZE, ROW_PER_WARP, float><<<dimGrid, dimBlock>>>(d_A, d_x, d_y, M, N);
        CHECK_CUDA(hipEventRecord(start));
        for (int run = 0; run < nIter; run++)
        {
            Sgemv_kernel_n16<WARP_SIZE, ROW_PER_WARP, float><<<dimGrid, dimBlock>>>(d_A, d_x, d_y, M, N);
            hipError_t err = hipGetLastError();
            if (err != hipSuccess)
            {
                printf("CUDA Error: %s\n", hipGetErrorString(err));
            }
        }
        CHECK_CUDA(hipEventRecord(stop));
        CHECK_CUDA(hipEventSynchronize(stop));
        CHECK_CUDA(hipEventElapsedTime(&gpu_time, start, stop));
        CHECK_CUDA(hipMemcpy(h_y_gpu, d_y, bytes_y, hipMemcpyDeviceToHost));
        printf("GPU time: %.5f seconds\n", gpu_time / nIter);
    }
    else if(N == 32) {
        dim3 dimGrid(M);  
        dim3 dimBlock(WARP_SIZE); 
        Sgemv_kernel_n32<WARP_SIZE, float><<<dimGrid, dimBlock>>>(d_A, d_x, d_y, M, N);
        CHECK_CUDA(hipEventRecord(start));
        for (int run = 0; run < nIter; run++)
        {
            Sgemv_kernel_n32<WARP_SIZE, float><<<dimGrid, dimBlock>>>(d_A, d_x, d_y, M, N);
            hipError_t err = hipGetLastError();
            if (err != hipSuccess)
            {
                printf("CUDA Error: %s\n", hipGetErrorString(err));
            }
        }
        CHECK_CUDA(hipEventRecord(stop));
        CHECK_CUDA(hipEventSynchronize(stop));
        CHECK_CUDA(hipEventElapsedTime(&gpu_time, start, stop));
        CHECK_CUDA(hipMemcpy(h_y_gpu, d_y, bytes_y, hipMemcpyDeviceToHost));
        printf("GPU time: %.5f seconds\n", gpu_time / nIter);
    }
    else if(N >= 128) {
        dim3 dimGrid(M);  
        dim3 dimBlock(WARP_SIZE); 
        Sgemv_kernel_n128<WARP_SIZE, float><<<dimGrid, dimBlock>>>(d_A, d_x, d_y, M, N);
        CHECK_CUDA(hipEventRecord(start));
        for (int run = 0; run < nIter; run++)
        {
            Sgemv_kernel_n128<WARP_SIZE, float><<<dimGrid, dimBlock>>>(d_A, d_x, d_y, M, N);
            hipError_t err = hipGetLastError();
            if (err != hipSuccess)
            {
                printf("CUDA Error: %s\n", hipGetErrorString(err));
            }
        }
        CHECK_CUDA(hipEventRecord(stop));
        CHECK_CUDA(hipEventSynchronize(stop));
        CHECK_CUDA(hipEventElapsedTime(&gpu_time, start, stop));
        CHECK_CUDA(hipMemcpy(h_y_gpu, d_y, bytes_y, hipMemcpyDeviceToHost));
        printf("GPU time: %.5f seconds\n", gpu_time / nIter);
    }
   
    // for (int i = 0; i < M; ++i) {
    //     std::cout << h_y_gpu[i] << " ";
    // }
    // std::cout << std::endl;
    // cublas
    hipblasHandle_t blas_handle;
    hipblasCreate(&blas_handle);
    float alpha = 1.0;
    float beta = 0;
    float api_time = 0;
    CHECK_CUDA(hipMemcpy(d_y, h_y, bytes_y, hipMemcpyHostToDevice));
    hipblasSgemv(blas_handle, HIPBLAS_OP_T, N, M, &alpha, d_A, N, d_x, 1, &beta, d_y, 1);
    CHECK_CUDA(hipEventRecord(start));
    for (int run = 0; run < nIter; run++)
    {
        hipblasSgemv(blas_handle, HIPBLAS_OP_T, N, M, &alpha, d_A, N, d_x, 1, &beta, d_y, 1);
    }
    CHECK_CUDA(hipEventRecord(stop));
    CHECK_CUDA(hipEventSynchronize(stop));
    CHECK_CUDA(hipEventElapsedTime(&api_time, start, stop));
    printf("API time: %.5f seconds\n", api_time / nIter);
    CHECK_CUDA(hipMemcpy(h_y_api, d_y, bytes_y, hipMemcpyDeviceToHost));
    hipblasDestroy(blas_handle);
    // for (int i = 0; i < M; ++i) {
    //     std::cout << h_y_api[i] << " ";
    // }
    // std::cout << std::endl;
    bool correct = true;
    correct = checkAnswer(h_y_api, h_y_cpu, M, 1);
    printf("API vs CPU %s\n", correct ? "Result= PASS" : "Result= FAIL");
    correct = checkAnswer(h_y_gpu, h_y_cpu, M, 1);
    printf("GPU vs CPU %s\n", correct ? "Result= PASS" : "Result= FAIL");
    // Free Memory
    hipFree(d_A);
    hipFree(d_x);
    hipFree(d_y);

    free(h_A);
    free(h_x);
    free(h_y);
    free(h_y_api);
    free(h_y_cpu);
    free(h_y_gpu);
}